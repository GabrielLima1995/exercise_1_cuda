#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//
//Print value of threadIdx,blockIdx, gridDim variables for 3D  grid which has 4 threads in all X, Y and Z 
// dimention and thread block size will be 2 threads in each dimention. 


// Kernell function 

__global__ void print_variables()
{
	printf("threadIdx : %d blockIdx : %d gridDim : %d \n",
		threadIdx.x, blockIdx.x, gridDim.x);

}

// Host function 

int main()
{	
	int x, y, z;

	x = 4;
	y = 4;
	z = 4;
	
	dim3 grid(2,2,2);
	dim3 block(x/grid.x,y/grid.y,z/grid.z);

	print_variables << <grid,block >> > ();

	return 0;
}

	